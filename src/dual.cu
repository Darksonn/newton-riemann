#include "hip/hip_runtime.h"

/*
  Class for dual numbers.
  Dual numbers exhibit the following property for any analytic function f:
    f(dual(x, 1)) = dual(f(x), f'(x))
  This makes them useful for calculating the value and derivative of a function.
 */
template <typename T>
class dual {
  public:
    T a, b;
    typedef T value_type;

    inline __host__ __device__
    dual(const T & _a = T(), const T & _b = T()) {
      a = _a;
      b = _b;
    }

    inline __host__ __device__
    dual<T> operator+(const dual<T> that) const {
      dual<T> out;
      out.a = a + that.a;
      out.b = b + that.b;
      return out;
    }
    inline __host__ __device__
    dual<T> operator-() const {
      dual<T> out;
      out.a = -a;
      out.b = -b;
      return out;
    }
    inline __host__ __device__
    dual<T> operator-(const dual<T> that) const {
      return operator+(-that);
    }
    inline __host__ __device__
    dual<T> operator*(const dual<T> that) const {
      dual<T> out;
      out.a = a * that.a;
      out.b = a * that.b + b * that.a;
      return out;
    }
    inline __host__ __device__
    dual<T> inverse() const {
      dual<T> out;
      out.a = T(1)/a;
      out.b = -b / (a*a);
      return out;
    }
    inline __host__ __device__
    dual<T> operator/(const dual<T> that) const {
      return operator*(that.inverse());
    }

};

template <typename T>
inline __host__ __device__
dual<T> dpow(dual<T> a, dual<T> b) {
  T one = T(1);
  T zero = T();
  dual<T> out;
  out.a = thrust::pow(a.a, b.a);
  out.b = b.a * a.b * thrust::pow(a.a, b.a-one);
  if (b.b != zero) out.b += b.b * out.a * thrust::log(a.a);
  return out;
}
template <typename T>
inline __host__ __device__
dual<T> dpown(dual<T> a, unsigned int n) {
  if (n == 0) return T(1);
  if (n == 1) return a;
  if (n % 2 == 0) return dpown(a*a, n / 2);
  return a * dpown(a*a, (n-1) / 2);
}
template <typename T>
inline __host__ __device__
dual<T> dlog(dual<T> x) {
  dual<T> out;
  out.a = thrust::log(x.a);
  out.b = x.b / x.a;
  return out;
}
template <typename T>
inline __host__ __device__
dual<T> dexp(dual<T> x) {
  dual<T> out;
  out.a = thrust::exp(x.a);
  out.b = x.b * out.a;
  return out;
}
template <typename T>
inline __host__ __device__
dual<T> dsin(dual<T> x) {
  dual<T> out;
  out.a = thrust::sin(x.a);
  out.b = x.b * thrust::cos(x.a);
  return out;
}
template <typename T>
inline __host__ __device__
dual<T> dcos(dual<T> x) {
  dual<T> out;
  out.a = thrust::cos(x.a);
  out.b = -x.b * thrust::sin(x.a);
  return out;
}
template <typename T>
inline __host__ __device__
dual<T> dtan(dual<T> x) {
  T one = T(1);
  T two = T(2);
  dual<T> out;
  out.a = thrust::tan(x.a);
  out.b = x.b * two / (cos(two*x.a)+one);
  return out;
}


