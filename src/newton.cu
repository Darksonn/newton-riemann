#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <stdint.h>

#include "gpu.h"
#include "color.h"

#define LOOP_COUNT 30000
#define GPU_ARRAYS 16
#define NEWTON_ITERS 2048

inline size_t len_from(unsigned int pixel_one) {
  unsigned int remaining = IMGW*IMGH - pixel_one;
  if (remaining > LOOP_COUNT) return LOOP_COUNT;
  return remaining;
}

void write_data(thrust::host_vector<struct rgb>& H, size_t len, unsigned int pixel_one) {
  for (size_t i = 0; i < len; ++i) {
    const unsigned int p = pixel_one + i;
    const unsigned int x = p % IMGW;
    const unsigned int y = p / IMGW;
    struct rgb c = H[i];
    printf("%u %u %d %d %d\n", x, y, (int) c.r, (int) c.g, (int) c.b);
  }
}

// the following three functions parse the time command-line parameter to a
// complex using recursive descent
f parse_float(char **str) {
  char *p = *str;
  bool exp_last = false;
  while (isdigit(*p) || *p == '.' || *p == 'e' || *p == 'E') {
loop_again:
    exp_last = *p == 'e' || *p == 'E';
    ++p;
  }
  if (exp_last && (*p == '-' || *p == '+')) goto loop_again;
  char c = *p;
  *p = 0;
  f res;
  sscanf(*str, "%" PRIf, &res);
  *p = c;
  *str = p;
  return res;
}
thrust::complex<f> parse_term(char **str) {
  thrust::complex<f> unit(1), i(0, 1);
  while (**str == 'i' || **str == 'j') {
    (*str)++;
    unit *= i;
  }
  f abs(parse_float(str));
  while (**str == 'i' || **str == 'j') {
    (*str)++;
    unit *= i;
  }
  return thrust::complex<f>(abs) * unit;
}
thrust::complex<f> parse_complex(char *str) {
  thrust::complex<f> final(0);
  while (*str) {
    thrust::complex<f> unit(1);
    if (*str == '-') unit = -1;
    if (*str == '-' || *str == '+') ++str;
    thrust::complex<f> term(parse_term(&str));
    final += unit * term;
  }
  return final;
}

int main(int argc, char **argv) {

  thrust::complex<f> t(0);
  if (argc > 1) {
    t = parse_complex(argv[1]);
  }

  thrust::device_vector<struct rgb> D[GPU_ARRAYS];
  thrust::host_vector<struct rgb> H(LOOP_COUNT);
  size_t lens[GPU_ARRAYS];
  unsigned int po[GPU_ARRAYS];
  memset(lens, 0, sizeof(size_t) * GPU_ARRAYS);
  for (int i = 0; i < GPU_ARRAYS; ++i) {
    D[i].resize(LOOP_COUNT);
  }

  int gpu = 0;
  uint32_t written = 0;
  unsigned int next_po = 0;
  while (1) {
    if (lens[gpu]) {
      H = D[gpu];
      write_data(H, lens[gpu], po[gpu]);
      written += lens[gpu];
      lens[gpu] = 0;
    }
    size_t len = len_from(next_po);
    if (len == 0) goto print;
    lens[gpu] = len;
    po[gpu] = next_po;
    thrust::counting_iterator<unsigned int> counter(next_po);
    newton_fast(NEWTON_ITERS, t, len, counter, D[gpu]);
    gpu = (gpu + 1) % GPU_ARRAYS;
    next_po += len;
  }
print:
  for (gpu = 0; gpu < GPU_ARRAYS; ++gpu) {
    if (lens[gpu]) {
      H = D[gpu];
      write_data(H, lens[gpu], po[gpu]);
      written += lens[gpu];
      lens[gpu] = 0;
    }
  }

  return 0;
}


