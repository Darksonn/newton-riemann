#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/complex.h>

#include "gpu.h"
#include "dual.cu"
#include "color.h"

typedef thrust::complex<f> complex;

// increase this value to make the image brighter
#define BRIGHTNESS_EPSILON 0.1

struct newton_iteration {
  const unsigned int iter;
  const dual<complex> t;

  newton_iteration(unsigned int _iter, complex _t) : iter(_iter), t(_t) {}

  __host__ __device__
  struct rgb operator()(const unsigned int& p) const {
    const unsigned int x = p % IMGW;
    const unsigned int y = p / IMGW;
    const f longitude = x * 2 * M_PI / (f) IMGW + M_PI;
    const f latitude  = y * M_PI / (2*(f) IMGH);
    complex z_ = thrust::exp(complex(0, longitude)) *
                 thrust::tan(complex(latitude, 0));
    for (unsigned int i = 0; i < iter; ++i) {
      const dual<complex> z(z_, 1);
      const dual<complex> fz =
#include "func.cu"
        ;
      z_ = z_ - fz.a / fz.b;
    }
    struct hsl hsl(thrust::arg(z_) / (2*M_PI), 1, 1 - 1 / (1 + thrust::abs(z_) * BRIGHTNESS_EPSILON));
    struct rgb rgb = HSLtoRGB(hsl);
    return rgb;
  }
};

void newton_fast(unsigned int iter, complex t, int len,
    thrust::counting_iterator<unsigned int>& X, thrust::device_vector<struct rgb>& Y) {
  thrust::transform(X, X + len, Y.begin(), newton_iteration(iter, t));
}

