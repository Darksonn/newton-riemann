#include "hip/hip_runtime.h"

#include "color.h"

__host__ __device__
inline bool is_weird(float f) {
  return isnan(f) || isinf(f);
}

__host__ __device__
struct rgb HSVtoRGB(struct hsv hsv) {
  struct rgb black;
  black.r = black.g = black.b = 0;
  if (is_weird(hsv.h) || is_weird(hsv.s) || is_weird(hsv.v)) return black;
  uint8_t r,g,b;
  float hue = hsv.h;
  float saturation = hsv.s;
  float brightness = hsv.v;
  if (saturation == 0) {
    r = g = b = (int) (brightness * 255.0f + 0.5f);
  } else {
    float h = (hue - (float)floor(hue)) * 6.0f;
    float f = h - (float)floor(h);
    float p = brightness * (1.0f - saturation);
    float q = brightness * (1.0f - saturation * f);
    float t = brightness * (1.0f - (saturation * (1.0f - f)));
    switch ((int) h) {
      case 0:
        r = (int) (brightness * 255.0f + 0.5f);
        g = (int) (t * 255.0f + 0.5f);
        b = (int) (p * 255.0f + 0.5f);
        break;
      case 1:
        r = (int) (q * 255.0f + 0.5f);
        g = (int) (brightness * 255.0f + 0.5f);
        b = (int) (p * 255.0f + 0.5f);
        break;
      case 2:
        r = (int) (p * 255.0f + 0.5f);
        g = (int) (brightness * 255.0f + 0.5f);
        b = (int) (t * 255.0f + 0.5f);
        break;
      case 3:
        r = (int) (p * 255.0f + 0.5f);
        g = (int) (q * 255.0f + 0.5f);
        b = (int) (brightness * 255.0f + 0.5f);
        break;
      case 4:
        r = (int) (t * 255.0f + 0.5f);
        g = (int) (p * 255.0f + 0.5f);
        b = (int) (brightness * 255.0f + 0.5f);
        break;
      case 5:
        r = (int) (brightness * 255.0f + 0.5f);
        g = (int) (p * 255.0f + 0.5f);
        b = (int) (q * 255.0f + 0.5f);
        break;
    }
  }
  struct rgb rgb;
  rgb.r = r;
  rgb.g = g;
  rgb.b = b;
  return rgb;
}
__host__ __device__
static float hue2rgb(float p, float q, float t) {
  if(t < 0) t += 1;
  if(t > 1) t -= 1;
  if(t < 1./6.) return p + (q - p) * 6 * t;
  if(t < 1./2.) return q;
  if(t < 2./3.) return p + (q - p) * (2./3. - t) * 6;
  return p;
}
__host__ __device__
struct rgb HSLtoRGB(struct hsl hsl) {
  float r, g, b;
  float h = hsl.h, s = hsl.s, l = hsl.l;

  if(s == 0){
    r = g = b = l; // achromatic
  }else{
    float q = l < 0.5 ? l * (1 + s) : l + s - l * s;
    float p = 2 * l - q;
    r = hue2rgb(p, q, h + 1./3.);
    g = hue2rgb(p, q, h);
    b = hue2rgb(p, q, h - 1./3.);
  }

  struct rgb rgb;
  rgb.r = uint8_t(r * 255);
  rgb.g = uint8_t(g * 255);
  rgb.b = uint8_t(b * 255);
  return rgb;
}

