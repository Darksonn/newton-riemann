#include "hip/hip_runtime.h"

template <typename T>
class dual {
  public:
    T a, b;
    typedef T value_type;

    inline __host__ __device__
    dual(const T & _a = T(), const T & _b = T()) {
      a = _a;
      b = _b;
    }

    inline __host__ __device__
    dual<T> operator+(const dual<T> that) {
      dual<T> out;
      out.a = a + that.a;
      out.b = b + that.b;
      return out;
    }
    inline __host__ __device__
    dual<T> operator-() {
      dual<T> out;
      out.a = -a;
      out.b = -b;
      return out;
    }
    inline __host__ __device__
    dual<T> operator-(dual<T> that) {
      return operator+(-that);
    }
    inline __host__ __device__
    dual<T> operator*(dual<T> that) {
      dual<T> out;
      out.a = a * that.a;
      out.b = a * that.b + b * that.a;
      return out;
    }
    inline __host__ __device__
    dual<T> inverse() {
      dual<T> out;
      out.a = T(1)/a;
      out.b = -b / (a*a);
      return out;
    }
    inline __host__ __device__
    dual<T> operator/(dual<T> that) {
      return operator*(that.inverse());
    }

};

template <typename T>
inline __host__ __device__
dual<T> dpow(dual<T> a, dual<T> b) {
  T one = T(1);
  T zero = T();
  dual<T> out;
  out.a = pow(a.a, b.a);
  out.b = b.a * a.b * pow(a.a, b.a-one);
  if (b.b != zero) out.b += b.b * out.a * log(a.a);
  return out;
}
template <typename T>
inline __host__ __device__
dual<T> dpown(dual<T> a, unsigned int n) {
  if (n == 0) return T(1);
  if (n == 1) return a;
  if (n % 2 == 0) return dpown(a*a, n / 2);
  return a * dpown(a*a, (n-1) / 2);
}
template <typename T>
inline __host__ __device__
dual<T> dlog(dual<T> x) {
  dual<T> out;
  out.a = log(x.a);
  out.b = x.b / x.a;
  return out;
}
template <typename T>
inline __host__ __device__
dual<T> dexp(dual<T> x) {
  dual<T> out;
  out.a = thrust::exp(x.a);
  out.b = x.b * out.a;
  return out;
}
template <typename T>
inline __host__ __device__
dual<T> dsin(dual<T> x) {
  dual<T> out;
  out.a = thrust::sin(x.a);
  out.b = x.b * thrust::cos(x.a);
  return out;
}
template <typename T>
inline __host__ __device__
dual<T> dcos(dual<T> x) {
  dual<T> out;
  out.a = thrust::cos(x.a);
  out.b = -x.b * thrust::sin(x.a);
  return out;
}
template <typename T>
inline __host__ __device__
dual<T> dtan(dual<T> x) {
  T one = T(1);
  T two = T(2);
  dual<T> out;
  out.a = thrust::tan(x.a);
  out.b = x.b * two / (cos(two*x.a)+one);
  return out;
}


